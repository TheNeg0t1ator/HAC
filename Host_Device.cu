#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


#define ARRAY_SIZE 256
#define NUM_BLOCKS  4
#define THREADS_PER_BLOCK 64

__global__ void negate(int *d_a)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    d_a[idx] = -1 * d_a[idx];
}

__global__ void negate_multiblock(int *d_a)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    d_a[idx] = -1 * d_a[idx];
}

int main(int argc, char *argv[])
{
    int *h_a, *h_out;
    int *d_a;

    int i;
    size_t siz_b = ARRAY_SIZE * sizeof(int);
    h_a = (int *) malloc(siz_b);
    h_out = (int *) malloc(siz_b);

    hipMalloc((void **)&d_a, siz_b);

    for (i = 0; i < ARRAY_SIZE; i++) {
        h_a[i] = i;
        h_out[i] = 0;
    }

    hipMemcpy(d_a, h_a, siz_b, hipMemcpyHostToDevice);

    dim3 blocksPerGrid(NUM_BLOCKS);
    dim3 threadsPerBlock(THREADS_PER_BLOCK);

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start measuring time
    hipEventRecord(start, 0);

    // Call the kerneln
    //negate<<<blocksPerGrid, threadsPerBlock>>>(d_a); // 0.173952 ms
    negate_multiblock<<<blocksPerGrid, threadsPerBlock>>>(d_a); // 0.176128 ms

    // Stop measuring time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed Time for negate kernel: %.6f ms\n", elapsedTime);

    hipMemcpy(h_out, d_a, siz_b, hipMemcpyDeviceToHost);

    printf("Results: ");
    for (i = 0; i < ARRAY_SIZE; i++) {
        printf("%d, ", h_out[i]);
    }
    printf("\n\n");

    hipFree(d_a);

    free(h_a);
    free(h_out);

    return 0;
}