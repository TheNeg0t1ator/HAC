#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" 
﻿
#define ARRAY_SIZE 256
#define NUM_BLOCKS  1
#define THREADS_PER_BLOCK 256
 
__global__ void negate(int *d_a)
{
 // CODE_1
}
 
__global__ void negate_multiblock(int *d_a)
{
 // CODE_2
}
 
int main(int argc, char *argv[])
{
    int *h_a, *h_out;
    int *d_a;
 
    int i;
    size_t siz_b = ARRAY_SIZE * sizeof(int);
    h_a = (int *) malloc(siz_b);
    h_out = (int *) malloc(siz_b);
 
    hipMalloc( );
 
    for (i = 0; i < ARRAY_SIZE; i++) {
        h_a[i] = i;
        h_out[i] = 0;
    }
 
    hipMemcpy( );
 
    //dim3 blocksPerGrid( ); 
    //dim3 threadsPerBlock( );
    negate<<< , >>>( );
    //negate_multiblock<<<,>>>();
    hipDeviceSynchronize();
 
    hipMemcpy( );
 
    printf("Results: ");
    for (i = 0; i < ARRAY_SIZE; i++) {
      printf("%d, ", h_out[i]);
    }
    printf("\n\n");
 
    hipFree( );
 
    free(h_a);
    free(h_out);
 
    return 0;
}